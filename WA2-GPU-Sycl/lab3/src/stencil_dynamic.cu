#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#define NROWS 1024        
#define NCOLS 1024        
#define STENCIL_RADIUS 1  

#define BLOCK_SIZE 256

// Macro for checking CUDA errors.
#define CHECK_CUDA(call) {                                    \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
        std::cerr << "CUDA error in " << __FILE__            \
                  << " at line " << __LINE__ << ": "          \
                  << hipGetErrorString(err) << std::endl;    \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
}

// Child kernel: processes one row using a 3-point stencil
__global__ void stencilKernel(const float* input, float* output, int ncols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ncols) {
        float sum = 0.0f;
        // Apply stencil to current element and its neighbors
        for (int r = -STENCIL_RADIUS; r <= STENCIL_RADIUS; r++) {
            int neighbor_col = col + r;
            // Check boundary conditions
            if (neighbor_col >= 0 && neighbor_col < ncols) {
                sum += input[neighbor_col];
            }
        }
        output[col] = sum;
    }
}

// Parent kernel: each thread handles one row and launches a child kernel
__global__ void parentKernel(const float* d_input, float* d_output, int nrows, int ncols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nrows) {
        // TODO: configure child kernel launch parameters.
        int blockSizeChild = BLOCK_SIZE;
        int gridSizeChild = (ncols + blockSizeChild - 1) / blockSizeChild;

        // TODO: launch the child kernel to process a row
        stencilKernel<<<gridSizeChild, blockSizeChild>>>(
            &d_input[row * ncols], 
            &d_output[row * ncols], 
            ncols
        );

        // Wait for the child kernel to finish.
        hipDeviceSynchronize();
    }
}

int main() {
    size_t matrixSize = NROWS * NCOLS * sizeof(float);
    float* h_input  = new float[NROWS * NCOLS];
    float* h_output = new float[NROWS * NCOLS];

    for (int i = 0; i < NROWS * NCOLS; i++) {
        h_input[i] = 1.0f;
    }

    // TODO: allocate device memory
    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc((void**)&d_input, matrixSize));
    CHECK_CUDA(hipMalloc((void**)&d_output, matrixSize));

    // TODO: copy input matrix from host to device
    CHECK_CUDA(hipMemcpy(d_input, h_input, matrixSize, hipMemcpyHostToDevice));

    // Launch the parent kernel. Each thread in this kernel will launch a child kernel
    // to process one row
    int blockSizeParent = BLOCK_SIZE;
    int gridSizeParent = (NROWS + blockSizeParent - 1) / blockSizeParent;
    parentKernel<<<gridSizeParent, blockSizeParent>>>(d_input, d_output, NROWS, NCOLS);

    // TODO: copy the processed matrix back to host memory
    CHECK_CUDA(hipMemcpy(h_output, d_output, matrixSize, hipMemcpyDeviceToHost));
    
    std::cout << "First row of result: ";
    for (int i = 0; i < NCOLS; i++) std::cout << h_output[i] << " ";
    std::cout << std::endl;

    // Clean up 
    delete[] h_input;
    delete[] h_output;
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));

    return 0;
}
